#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <ctime>
#include <stdarg.h>
#include <hip/hip_runtime_api.h>

using namespace std;

#define BUFSIZE 4096

void cuda_check(hipError_t err, const char *msg, ...) {
	if (err != hipSuccess) {
		char msg_buf[BUFSIZE + 1];
		va_list ap;
		va_start(ap, msg);
		vsnprintf(msg_buf, BUFSIZE, msg, ap);
		va_end(ap);
		msg_buf[BUFSIZE] = '\0';
		fprintf(stderr, "%s - error %d (%s)\n", msg_buf, err, hipGetErrorString(err));
		exit(1);
	}
}

__host__ __device__   // NVCC compiles this code for both device and host
int4 operator+(int4 const & a, int4 const & b)
{
  return make_int(a.x+b.x, a.y+b.y, a.z+b.z, a.w+b.w);
}

__global__
void vecinit(int * __restrict__ v1, int * __restrict__ v2, int nels)
{
  // threadIdx is a int3 index (xyz). Local workitem index <=> get_local_id()
  // blockIdx is the workgroup's index in the launch grid. <=> get_group_id()
  // blockDim is the workgroup dimension                   <=> get_group_size()
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  v1[i] = i;
	v2[i] = nels - i;
}

// Performance normale
__global__
void vecsum(int *__restrict__ vsum, const int *__restrict__ v1, const int *__restrict__ v2, int nels)
{
	const int i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i>=nels) return;

	vsum[i] = v1[i] + v2[i];
}


void verify(const int *vsum, int nels)
{
	for (int i = 0; i < nels; ++i) 
		if (vsum[i] != nels) 
		{
			fprintf(stderr, "mismatch @ %d : %d != %d\n", i, vsum[i], nels);
			exit(3);
		}
}


int main(int argc, char *argv[])
{
	if (argc <= 1) 
	{
		fprintf(stderr, "specify number of elements\n");
		exit(1);
	}

	const int nels = atoi(argv[1]);
  
  int *d_v1 = NULL, *d_v2 = NULL, *d_vsum = NULL;

  size_t memsize = nels*sizeof(*d_v1);
  
  hipError_t err;

  err = hipMalloc(&d_v1, memsize); // hipMalloc returns a device pointer
  cuda_check(err,"alloc v1");
  err = hipMalloc(&d_v2, memsize);
  cuda_check(err,"alloc v2");
  err = hipMalloc(&d_vsum, memsize);
  cuda_check(err,"alloc vsum");
  
  int blockSize = 256;
  int numBlocks = (nels + blockSize - 1)/blockSize;

  printf("%d blocchi di %d threads\n",numBlocks,blockSize);

  hipEvent_t pre_init, post_init;
  hipEvent_t pre_sum, post_sum;
  hipEvent_t pre_copy, post_copy;

  err = hipEventCreate(&pre_init, 0);
  cuda_check(err, "pre_init event create");
  err = hipEventCreate(&post_init, 0);
  cuda_check(err, "post_init event create");
  err = hipEventCreate(&pre_sum, 0);
  cuda_check(err, "pre_sum event create");
  err = hipEventCreate(&post_sum, 0);
  cuda_check(err, "post_sum event create");
  err = hipEventCreate(&pre_copy, 0);
  cuda_check(err, "post_copy event create");
  err = hipEventCreate(&post_copy, 0);
  cuda_check(err, "post_copy event create");

  err = hipEventRecord(pre_init);
  cuda_check(err, "pre_init record");
  vecinit<<< numBlocks, blockSize >>>(d_v1,d_v2,nels);
  err = hipEventRecord(post_init);
  cuda_check(err, "post_init record");

  err = hipEventRecord(pre_sum);
  cuda_check(err, "pre_sum record");
  vecsum<<< numBlocks, blockSize >>>(d_vsum,d_v1,d_v2,nels);
  err = hipEventRecord(post_sum);
  cuda_check(err, "post_sum record");

  int * h_vsum;
  err = hipHostAlloc(&h_vsum, memsize, hipHostMallocPortable); //DMA


  if (!h_vsum)
  {
    fprintf(stderr, "out of memory on host!\n");
    exit(1);
  }

  err = hipEventRecord(pre_copy);
  cuda_check(err, "pre_copy record");
  err = hipMemcpy(h_vsum,d_vsum,memsize, hipMemcpyDeviceToHost);
  cuda_check(err, "copy vsum");
  err = hipEventRecord(post_copy);
  cuda_check(err, "post_copy record");

	verify(h_vsum, nels);

  hipEventSynchronize(post_copy);
  cuda_check(err, "sync post_copy");

  float init_time, sum_time, copy_time;
  err = hipEventElapsedTime(&init_time, pre_init, post_init);  //in ms
  cuda_check(err, "get init time");
  err = hipEventElapsedTime(&sum_time, pre_sum, post_sum);  //in ms
  cuda_check(err, "get init time");
  err = hipEventElapsedTime(&copy_time, pre_copy, post_copy);  //in ms
  cuda_check(err, "get init time");

  printf("init: %d els in &6.4gms: %6.4gGB/s, %6.4gGE/s\n", nels, init_time,2*memsize/init_time/1.0e6, nels/init_time/1.0e6);
  printf("sum: %d els in &6.4gms: %6.4gGB/s, %6.4gGE/s\n", nels, sum_time,3*memsize/sum_time/1.0e6, nels/sum_time/1.0e6);
  printf("copy: %d els in &6.4gms: %6.4gGB/s, %6.4gGE/s\n", nels, copy_time,memsize/copy_time/1.0e6, nels/copy_time/1.0e6);

  hipEventDestroy(pre_init);
  hipEventDestroy(post_init);
  hipEventDestroy(pre_sum);
  hipEventDestroy(post_sum);
  hipEventDestroy(pre_copy);
  hipEventDestroy(post_copy);

  
  hipHostFree(h_vsum);     h_vsum = NULL;
  hipFree(d_vsum); d_vsum = NULL;
  hipFree(d_v2);   d_v2 = NULL;
  hipFree(d_v1);   d_v1 = NULL;
}