#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <ctime>
#include <stdarg.h>
#include <hip/hip_runtime_api.h>

using namespace std;

#define BUFSIZE 4096

void cuda_check(hipError_t err, const char *msg, ...) {
	if (err != hipSuccess) {
		char msg_buf[BUFSIZE + 1];
		va_list ap;
		va_start(ap, msg);
		vsnprintf(msg_buf, BUFSIZE, msg, ap);
		va_end(ap);
		msg_buf[BUFSIZE] = '\0';
		fprintf(stderr, "%s - error %d (%s)\n", msg_buf, err, hipGetErrorString(err));
		exit(1);
	}
}

__host__ __device__   // NVCC compiles this code for both device and host
int4 operator+(int4 const & a, int4 const & b)
{
  return make_int(a.x+b.x, a.y+b.y, a.z+b.z, a.w+b.w);
}

__global__
void vecinit(int * __restrict__ v1, int * __restrict__ v2, int nels)
{
  // threadIdx is a int3 index (xyz). Local workitem index <=> get_local_id()
  // blockIdx is the workgroup's index in the launch grid. <=> get_group_id()
  // blockDim is the workgroup dimensione                  <=> get_group_size()
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  v1[i] = i;
	v2[i] = nels - i;
}

// Performance normale
__global__
void vecsum(int *__restrict__ vsum, const int *__restrict__ v1, const int *__restrict__ v2, int nels)
{
	const int i = get_global_id(0);

    if(i>=nels)  return;

	vsum[i] = v1[i] + v2[i];
}


void verify(const int *vsum, int nels)
{
	for (int i = 0; i < nels; ++i) 
		if (vsum[i] != nels) 
		{
			fprintf(stderr, "mismatch @ %d : %d != %d\n", i, vsum[i], nels);
			exit(3);
		}
}


int main(int argc, char *argv[])
{
	if (argc <= 1) 
	{
		fprintf(stderr, "specify number of elements\n");
		exit(1);
	}

	const int nels = atoi(argv[1]);
  
  int d_v1 = NULL, d_v2 = NULL, d_vsum = NULL;

  size_t memsize = nels*sizeof(*d_v1);
  
  hipError_t err;

  err = (int*)hipMalloc(&d_v1, memsize); // hipMalloc returns a device pointer
  cuda_check(err,"alloc v1");
  err = (int*)hipMalloc(&d_v2, memsize);
  cuda_check(err,"alloc v2");
  err = (int*)hipMalloc(&d_vsum, memsize);
  cuda_check(err,"alloc vsum");
  
  int blockSize = 256;
  int numBlocks = (nels + blockSize - 1)/blockSize;

  vecinit<<< blockSize, numBlocks >>>(d_v1,d_v2,nels);
  
  vecsum<<< blockSize, numBlocks >>>(d_vsum,d_v1,d_v2,nels);
  
  int * h_vsum = (int*)malloc(memsize);

  if (!h_vsum)
  {
    fprintf(stderr, "out of memory on host!\n");
    exit(1);
  }

  err = hipMemcpy(h_vsum,d_vsum,memsize, hipMemcpyDeviceToHost);
  cuda_check(err "copy vsum");

	verify(h_vsum, nels);

  free(h_vsum);     h_vsum = NULL;
  hipFree(d_vsum); d_vsum = NULL;
  hipFree(d_v2);   d_v2 = NULL;
  hipFree(d_v1);   d_v1 = NULL;
}