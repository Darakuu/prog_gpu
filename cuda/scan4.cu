
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>
#include <ctime>
#include <stdarg.h>
#include <hip/hip_runtime_api.h>

#define BUFSIZE 4096

typedef unsigned int uint;

using namespace std;

void cuda_check(hipError_t err, const char *msg, ...) 
{
  if (err != hipSuccess) 
  {
		char msg_buf[BUFSIZE + 1];
		va_list ap;
		va_start(ap, msg);
		vsnprintf(msg_buf, BUFSIZE, msg, ap);
		va_end(ap);
		msg_buf[BUFSIZE] = '\0';
		fprintf(stderr, "%s - error %d (%s)\n", msg_buf, err, hipGetErrorString(err));
		exit(1);
	}
}

__host__ __device__   // NVCC compiles this code for both device and host
int4 operator+(int4 const & a, int4 const & b)
{
  return make_int4(a.x+b.x, a.y+b.y, a.z+b.z, a.w+b.w);
}

__host__ __device__   // NVCC compiles this code for both device and host
void operator+=(int4 & a, int b)
{
  a.x += b;
  a.y += b;
  a.z += b;
  a.w += b;
}

__device__
void microscan(int4 &q)
{
  q.y += q.x;
  q.z += q.y;
  q.w += q.z;
}

// a / b, rounding up
__host__ __device__
uint div_up(uint a, uint b) {return (a + b - 1)/b;}

// round a to the next multiple of b
__host__ __device__
uint round_mul_up(uint a, uint b) {return div_up(a, b)*b;}

__device__
int get_global_id() {return blockIdx.x * blockDim.x + threadIdx.x;}

__global__
void vecinit(int *out, int n)
{
	const int i = get_global_id();
	if (i < n) out[i] = (i+1);
}

// CUDA Shared Memory
extern __shared__ int lmem[];

// scan that accesses global memory directly
__device__
int scan_pass(int gi, int nels,
	 int * __restrict__ out,
	 const int * __restrict__ in,
	 int * __restrict__ lmem,
	 int corr)
{
	const uint li = threadIdx.x;
	const uint lws = blockDim.x;
	int acc = (gi < nels ? in[gi] : 0);

	uint write_mask = 1U;
	uint read_mask = ~0U;

	lmem[li] = acc;
	while (write_mask < lws) 
  {
		__syncthreads();
		if (li & write_mask) 
    {
			acc += lmem[(li & read_mask) - 1];
			lmem[li] = acc;
		}
		write_mask <<= 1;
		read_mask <<= 1;
	}

	acc += corr;
	if (gi < nels)
		out[gi] = acc;

  __syncthreads();
	corr += lmem[lws - 1];

	// ensure that lmem[i] on the next cycle from the last work-item does not
	// overwrite lmem[lws-1] before all other work-item read it
	__syncthreads();
	return corr;
}

// scan pass in which we pass the values that need to be scanned
__device__
int scan_pass(int gi, int global_nquarts, 
    int4 * __restrict__ out, 
    int4 &q, 
    int * __restrict__ lmem, 
    int corr)
{
	const uint li = threadIdx.x;
	const uint lws = blockDim.x;
	int acc = q.w;

	uint write_mask = 1U;
	uint read_mask = ~0U;

	lmem[li] = acc;
	while (write_mask < lws) 
  {
		__syncthreads();
		if (li & write_mask) 
    {
			acc += lmem[(li & read_mask) - 1];
			lmem[li] = acc;
		}
		write_mask <<= 1;
		read_mask <<= 1;
	}

  // complete work-group scan
  __syncthreads();
  if (li > 0)
    q += lmem[li-1];
  
  // sliding window correction
  q+= corr;

	if (gi < global_nquarts)
		out[gi] = q;

	corr += lmem[lws - 1];

  // ensure that lmem[i] on the next cycle
  // from the last work-item does not
	// overwrite lmem[lws-1] before all other work-item read it
	__syncthreads();
	return corr;
}

/* single-work-group version: used to scan the tails of the partial scans */
__global__
void scan1_lmem(int * __restrict__ out,
	const int * __restrict__ in,
	uint nels)
{
	const uint lws = blockDim.x;
	const uint limit = round_mul_up(nels, lws);
  uint gi = get_global_id();
  int corr = 0;
  
	while (gi < limit) // 4x4 loop
  {
    corr = scan_pass(gi,nels,out,in,lmem,corr);
    gi += lws;
	}
}

/* multi-work-group, vectorized (x4) version 
*  each work-item reads and writes a int4.
*  once a work-item is done, it save a single int.
*/
__global__
void scan4_lmem(int4 * __restrict__ out,
	int * __restrict__ tails,
	const int4 * __restrict__ in, 
	uint global_nquarts)
{
	const uint lws = blockDim.x;

	// number of quarts for the single work-group:
	// start by dividing the total number of elements by the number of groups,
	// rounding up
	uint local_nquarts = div_up(global_nquarts, gridDim.x);
	// round up to the next multiple of lws
	local_nquarts = round_mul_up(local_nquarts, lws);

	const uint begin = blockIdx.x*local_nquarts;
	const uint end = min(begin + local_nquarts, global_nquarts);
	const uint limit = round_mul_up(end, lws);
	int corr = 0;

	uint gi = begin + threadIdx.x;
	while (gi < limit)
  {
    int g0 = gi + 0*lws;
    int g1 = gi + 1*lws;
    int g2 = gi + 2*lws;
    int g3 = gi + 3*lws;

    int4 q0 = g0 < global_nquarts ? in[g0] : make_int4(0,0,0,0);
    int4 q1 = g1 < global_nquarts ? in[g1] : make_int4(0,0,0,0);
    int4 q2 = g2 < global_nquarts ? in[g2] : make_int4(0,0,0,0);
    int4 q3 = g3 < global_nquarts ? in[g3] : make_int4(0,0,0,0);

    microscan(q0);
    microscan(q1);
    microscan(q2);
    microscan(q3);

    corr = scan_pass(g0, global_nquarts, out, q0, lmem, corr);
    corr = scan_pass(g1, global_nquarts, out, q1, lmem, corr);
    corr = scan_pass(g2, global_nquarts, out, q2, lmem, corr);
    corr = scan_pass(g3, global_nquarts, out, q3, lmem, corr);
    
    gi += 4*lws;
    /*  
    ALTERNATIVE: exploit instruction level parameters (ILP).
    If the hardware allows it, these are not 4 sums but 2 parallelized sums.
    q.y += q.x
    q.w += q.z // these 2 are indipendent sums
    q.z += q.y
    q.w += q.y // idem
    */
	}

	if (threadIdx.x == 0)
		tails[blockIdx.x] = corr;
}

/* fixup the partial scans with the scanned tails */
__global__
void scanN_fixup(int4 * __restrict__ out,
	const int * __restrict__ tails,
	uint global_nquarts)
{
	if (blockIdx.x == 0) return;

	const uint lws = blockDim.x;

	// number of elements for the single work-group:
	// start by dividing the total number of elements by the number of groups,
	// rounding up
	uint local_nquarts = div_up(global_nquarts, gridDim.x);
	// round up to the next multiple of lws
	local_nquarts = round_mul_up(local_nquarts, lws);

	const uint begin = blockIdx.x*local_nquarts;
	const uint end = min(begin + local_nquarts, global_nquarts);
	const int corr = tails[blockIdx.x-1];

	uint gi = begin + threadIdx.x;
	while (gi < end) 
  {
		out[gi] += corr;
		gi += lws;
	}
}

void verify(const int *vsum, int nels)
{
	int scan = 0;
	for (int i = 0; i < nels; ++i) 
  {
		scan += (i+1);
		if (vsum[i] != scan) 
			fprintf(stderr, "mismatch @ %d : %d != %d\n", i, vsum[i], scan), exit(3);
	}
}


int main(int argc, char *argv[])
{
	if (argc <= 3)
		fprintf(stderr, "specify number of elements, lws, nwg\n"), exit(1);

  const int nels = atoi(argv[1]);
  const int lws = atoi(argv[2]);
  const int nwg = atoi(argv[3]);
  const size_t memsize = nels*sizeof(int);
  const size_t nwg_mem = nwg*sizeof(int);
 
  if (nels & 3) cuda_check(hipErrorInvalidValue, "nels must be multiple of 4");
  if (lws & (lws-1)) cuda_check(hipErrorInvalidValue, "lws"); // this should be invalid value, not invalid device(!!!)
  
  int *d_v1 = NULL, *d_v2 = NULL, *d_tails = NULL;
  
  hipError_t err;

  err = hipMalloc(&d_v1, memsize); // cudaMalloc returns a device pointer
  cuda_check(err,"alloc v1");
  err = hipMalloc(&d_v2, memsize);
  cuda_check(err,"alloc v2");
  err = hipMalloc(&d_tails, nwg_mem);
  cuda_check(err,"alloc vsum");

  hipEvent_t pre_init, post_init;
  hipEvent_t pre_scan1, post_scan1;
  hipEvent_t pre_scan_tails, post_scan_tails;
  hipEvent_t pre_fixup, post_fixup;
  hipEvent_t pre_copy, post_copy;

  err = hipEventCreateWithFlags(&pre_init, 0);
  cuda_check(err, "pre_init event create");
  err = hipEventCreateWithFlags(&post_init, 0);
  cuda_check(err, "post_init event create");
  
  err = hipEventCreateWithFlags(&pre_scan1, 0);
  cuda_check(err, "pre_scan1 event create");
  err = hipEventCreateWithFlags(&post_scan1, 0);
  cuda_check(err, "post_scan1 event create");
  
  err = hipEventCreateWithFlags(&pre_scan_tails, 0);
  cuda_check(err, "pre_scan_tails event create");
  err = hipEventCreateWithFlags(&post_scan_tails, 0);
  cuda_check(err, "post_scan_tails event create");
  
  err = hipEventCreateWithFlags(&pre_fixup, 0);
  cuda_check(err, "pre_fixup event create");
  err = hipEventCreateWithFlags(&post_fixup, 0);
  cuda_check(err, "post_fixup event create");

  err = hipEventCreateWithFlags(&pre_copy, 0);
  cuda_check(err, "pre_copy event create");
  err = hipEventCreateWithFlags(&post_copy, 0);
  cuda_check(err, "post_copy event create");

  err = hipEventRecord(pre_init);
  cuda_check(err, "pre_init record");
  vecinit<<< div_up(nels, lws), lws >>>(d_v1,nels);
  err = hipEventRecord(post_init);
  cuda_check(err, "post_init record");

  int nquarts = nels/4;
  err = hipEventRecord(pre_scan1);
  cuda_check(err, "pre_scan1 record");
  if (nwg>1)
    scan4_lmem<<< nwg, lws, lws*sizeof(int) >>>((int4*)d_v2, d_tails, (int4*)d_v1, nquarts);
  else
    scan1_lmem<<< 1, lws,lws*sizeof(int) >>>(d_v2, d_v1, nels);
  err = hipEventRecord(post_scan1);
  cuda_check(err, "post_scan1 record");

  if (nwg > 1)
  {
    err = hipEventRecord(pre_scan_tails);
    cuda_check(err, "record pre_scan_tails");
    scan1_lmem<<< 1, lws,lws*sizeof(int) >>>(d_tails, d_tails, nwg);
    err = hipEventRecord(post_scan_tails);
    cuda_check(err, "record post_scan_tails");

    err = hipEventRecord(pre_fixup);
    cuda_check(err, "record pre_fixup");
    scanN_fixup<<< nwg, lws >>>((int4*)d_v2, d_tails, nquarts);
    err = hipEventRecord(post_fixup);
    cuda_check(err, "record post_fixup");
  }

  int * h_v2;
  err = hipHostAlloc(&h_v2, memsize, hipHostMallocPortable); //DMA
  cuda_check(err, "host alloc");

  err = hipEventRecord(pre_copy);
  cuda_check(err, "pre_copy record");
  err = hipMemcpy(h_v2,d_v2,memsize, hipMemcpyDeviceToHost);
  cuda_check(err, "copy v2");
  err = hipEventRecord(post_copy);
  cuda_check(err, "post_copy record");

	verify(h_v2, nels);

  hipEventSynchronize(post_copy);
  cuda_check(err, "sync post_copy");

  float init_time, scan1_time, scan_tails_time, fixup_time, copy_time;
  float scan_time_total;
  err = hipEventElapsedTime(&init_time, pre_init, post_init);  //in ms
  cuda_check(err, "get init time");
  err = hipEventElapsedTime(&scan1_time, pre_scan1, post_scan1);  //in ms
  cuda_check(err, "get scan1 time");
  if (nwg > 1)
  {
    err = hipEventElapsedTime(&scan_tails_time, pre_scan_tails, post_scan_tails);  //in ms
    cuda_check(err, "get scan tails time");
    err = hipEventElapsedTime(&fixup_time, pre_fixup, post_fixup);  //in ms
    cuda_check(err, "get fixup time");
  }
  err = hipEventElapsedTime(&copy_time, pre_copy, post_copy);  //in ms
  cuda_check(err, "get copy time");
  err = hipEventElapsedTime(&scan_time_total, pre_scan1, nwg > 1 ? post_fixup : post_scan1);
  cuda_check(err, "get scan time total");

  printf("init: %d els in %6.4gms: %6.4gGB/s, %6.4gGE/s\n", nels, init_time, memsize/init_time/1.0e6, nels/init_time/1.0e6);
  printf("scan0:  %d els in %6.4gms: %6.4gGB/s, %6.4gGE/s\n", nels, scan1_time,2*memsize/scan1_time/1.0e6, nels/scan1_time/1.0e6);
  if(nwg>1)
  {
    printf("scan tails:  %d els in %6.4gms: %6.4gGB/s, %6.4gGE/s\n", nwg, scan_tails_time,2*nwg_mem/scan_tails_time/1.0e6, nwg/scan_tails_time/1.0e6);
    printf("scan fixup:  %d els in %6.4gms: %6.4gGB/s, %6.4gGE/s\n", nels, fixup_time,2*(memsize - lws*sizeof(int))/fixup_time/1.0e6, nels/fixup_time/1.0e6);
  }
  printf("copy: %d els in %6.4gms: %6.4gGB/s, %6.4gGE/s\n", nels, copy_time,memsize/copy_time/1.0e6, nels/copy_time/1.0e6);
  printf("scan total: %d els in %6.4gms: %6.4gGB/s, %6.4gGE/s\n", nels, scan_time_total,memsize/scan_time_total/1.0e6, nels/scan_time_total/1.0e6);

  hipEventDestroy(pre_init);
  hipEventDestroy(post_init);
  hipEventDestroy(pre_scan1);
  hipEventDestroy(post_scan1);
  hipEventDestroy(pre_scan_tails);
  hipEventDestroy(post_scan_tails);
  hipEventDestroy(pre_fixup);
  hipEventDestroy(post_fixup);
  hipEventDestroy(pre_copy);
  hipEventDestroy(post_copy);

  hipHostFree(h_v2);     h_v2 = NULL;
  hipFree(d_v2);   d_v2 = NULL;
  hipFree(d_tails); d_tails = NULL;
  hipFree(d_v1);   d_v1 = NULL;
}